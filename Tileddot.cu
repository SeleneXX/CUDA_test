
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include <sys/time.h> 
#include <unistd.h>
using namespace std;

#define BLOCK_SIZE 32
// Tileddot cuda function
__global__ void dot(const long* _A,const long* _B, long* _C, long _wa)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;


    int aBegin = _wa*(by*BLOCK_SIZE);
    int aEnd = aBegin + _wa - 1;
    int aStep = BLOCK_SIZE;//offsetA

    int bBegin = BLOCK_SIZE*bx;//B(bx,0)
    int bStep = BLOCK_SIZE*_wa;//offsetB

    float cSub = 0;
    for (int a = aBegin,b = bBegin; a <= aEnd; a += aStep,b += bStep)
    {
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[ty][tx] = _A[a + _wa*ty + tx];
        Bs[ty][tx] = _B[b + _wa*ty + tx];

        __syncthreads();


        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            cSub += As[ty][k]*Bs[k][tx];
        }

        __syncthreads();
    }


    int cIndex = (by*BLOCK_SIZE + ty)*_wa + (bx*BLOCK_SIZE + tx);
    _C[cIndex] = cSub;
}


int main()
{
    //six time will be GPU faster than CPU
    // number of time of teset, the max width and length of matrix will be 2^test_time
    int test_time = 8;
    // to store time
    struct timeval start;
    struct timeval end;
    long cpu_runtimes[test_time];
    long gpu_runtimes[test_time];
    // loop to test vector add
    for(int j = 0; j<test_time; j++){
        // adding the zeros after 0010 to get 2^test_time size number
        int N = 2 << j;
        // calculate size of matrix
        long nBytes = N*N*sizeof(long);
        // get host memory
        long *x, *y, *z;
        x = (long*)malloc(nBytes);
        y = (long*)malloc(nBytes);
        z = (long*)malloc(nBytes);

        // initialized data
        for (int i = 0; i < N*N; ++i)
        {
            x[i] = 1;
            y[i] = 2;
        }

        // get device memory
        long *d_x, *d_y, *d_z;
        hipMalloc((void**)&d_x, nBytes);
        hipMalloc((void**)&d_y, nBytes);
        hipMalloc((void**)&d_z, nBytes);

        int bs =32;
        if(N<32){
            bs=N;
        }
        // kernel configeration
        dim3 blockSize(bs,bs);
        dim3 gridSize(ceilf(N/(float)blockSize.x),ceilf(N/(float)blockSize.y) );

        //  CUP vector add
        gettimeofday(&start,NULL);
        for(int i = 0; i < N; i++){
            for(int k=0;k<N; k++){
                for(int h=0; h<N;h++){
                   z[i*N + h] += x[i*N +k]*y[k*N +h]; 
                }
            }   
        }
        gettimeofday(&end,NULL);
        // CPU Result examnation and store runtime
        cpu_runtimes[j] =  end.tv_usec - start.tv_usec;

        // GPU vector add 
        gettimeofday(&start,NULL);
        // run gpu function
        //  cpoy  data form host to device
        hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
        hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);

        dot <<< gridSize, blockSize >>>(d_x, d_y, d_z, N);

        // Synchronize GPU and get result
        hipDeviceSynchronize();
        hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyDeviceToHost);

        gettimeofday(&end,NULL);
        // GPU Result examnation and store runtime
        gpu_runtimes[j] =  end.tv_usec - start.tv_usec;
        // Free Memory
        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_z);
        free(x);
        free(y);
        free(z);

    }
    
    // print CPU runtime
    std::cout << "---- Matrix Tile Multiplication ----"<< std::endl ;
    std::cout << "Matrix width and legnth changes from 2^1 to 2^"<<test_time<< std::endl;;
    std::cout << "CPU runtimes: ";
    for(int j = 0; j<test_time; j++){
        std::cout <<  cpu_runtimes[j] << ", ";
    }
    std::cout << std::endl;
    // print GPU runtime
    std::cout << "GPU runtimes: ";
    for(int j = 0; j<test_time; j++){
        std::cout <<  gpu_runtimes[j] << ", ";
    }
    std::cout << std::endl;
   

    return 0;
}